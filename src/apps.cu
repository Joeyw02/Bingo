#include <iostream>
#include <cstdio>
#include <cstring>
#include <omp.h>
#include "apps.h"
#include "utils.cuh"
#include "test.cuh"
#include "graph.cuh"
using namespace std;
int instance(app a, utype utp, string DATASET)
{
    freopen(("../dataset/" + DATASET).c_str(), "r", stdin);
    cerr << "Graph dataset: " << DATASET << "." << endl;
    if (a == app::node2vec)
        NODE2VEC = 1;
    Timer TT;
    TT.restart();
    omp_set_num_threads(CPUTHD);
    loadGraph();
    Timer T;
    T.restart();
    if (utp == utype::M)
        BATCHSIZE >>= 1;
    for (int i = 0; i < 10; ++i)
    {
        if (utp == utype::M || utp == utype::I)
            insertGraph();
        if (utp == utype::M || utp == utype::D)
            deleteGraph();
        randomWalk(a);
    }
    double tt = T.duration();
    cerr << "Random walk in " << totalTime << " s." << endl;
    cerr << "Evaluation time: " << tt << " s." << endl;
    if (DETAIL)
        cerr << "Total time: " << TT.duration() << " s." << endl;
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    cerr << "Memory Consumption: " << ((total_mem - free_mem) / 1024. / 1024. - 4) / 1024. << " GB." << endl; // cerr<<xxx<<" "<<tt-totalTime-xxx<<endl;
    fclose(stdin);
    return 0;
}