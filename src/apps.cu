#include <iostream>
#include <cstdio>
#include <cstring>
#include <omp.h>
#include "apps.h"
#include "utils.cuh"
#include "test.cuh"
#include "graph.cuh"
using namespace std;
int instance(app a, utype utp, string DATASET)
{
    // freopen(("../dataset/" + DATASET).c_str(), "r", stdin);
    FILE* fd = freopen(("../dataset/" + DATASET).c_str(), "r", stdin);
    if (!fd) {
        perror(("Failed to open file: ../dataset/" + DATASET).c_str());
        exit(EXIT_FAILURE);
    }
    
    cerr << "Graph dataset: " << DATASET << "." << endl;
    if (a == app::node2vec)
        NODE2VEC = 1;
    Timer TT;
    TT.restart();
    omp_set_num_threads(CPUTHD);
    loadGraph();
    Timer T;
    T.restart();
    if (utp == utype::M)
        BATCHSIZE >>= 1;
    for (int i = 0; i < 10; ++i)
    {
        if (utp == utype::M || utp == utype::I)
            insertGraph();
        if (utp == utype::M || utp == utype::D)
            deleteGraph();
        randomWalk(a);
    }
    double tt = T.duration();
    cerr << "Random walk in " << totalTime << " s." << endl;
    cerr << "Evaluation time: " << tt << " s." << endl;
    if (DETAIL)
        cerr << "Total time: " << TT.duration() << " s." << endl;
    
    // todo: 计算真实采样索引空间大小
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    cerr << "Memory Consumption: " << ((total_mem - free_mem) / 1024. / 1024. - 4) / 1024. << " GB." << endl; // cerr<<xxx<<" "<<tt-totalTime-xxx<<endl;
    fclose(stdin);
    return 0;
}