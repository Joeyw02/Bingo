#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<cstring>
#include<omp.h>
#include"api.h"
#include"apps.h"
#include"gpuMem.cuh"
#include"operation.cuh"
#include"Bingo.cuh"
#include"utils.cuh"
using namespace std;
CPURand r;
int n,batchSize=100000;
int *d,*sizeManager;
vector<EdgeData>edgeData;
bool buffer=0;
NodeData *nd[GPUS],*ndD[GPUS];
void loadGraph(){    
    r.init();mp.init();
    int u,v;n=0;
    while(scanf("%d%d",&u,&v)!=EOF){
        EdgeData tmp;tmp.u=u+1,tmp.v=v+1;
        edgeData.push_back(tmp);
    }   
    cerr<<"Edge number: "<<edgeData.size()<<"."<<endl;
    for(int i=0;i<edgeData.size();++i)n=max(n,max(edgeData[i].u,edgeData[i].v));
    int amount=edgeData.size();cerr<<n<<endl;
    //batchSize=amount*ChangeR;
    Edges *edges=new Edges[amount];
    d=new int[n+1];
    memset(d,0,sizeof(int)*(n+1));
    for(int i=0;i<amount;++i){
        EdgeData *tmp=&edgeData[i];
        tmp->nodeIdu=d[tmp->u]++;
    }
    #pragma omp parallel for
    for(int i=0;i<amount;++i){
        int u=edgeData[i].u,v=edgeData[i].v;
        edges[i]=(Edges){u,v,(unsigned)d[v]};
    }
    build(edges,amount,nd,ndD,d,&sizeManager,n);
    delete[] edges;
}
void insertGraph(){
    Edges *edges=new Edges[batchSize];
    int lastAmount=edgeData.size();
    for(int i=0;i<batchSize;++i){
        int pos=r.rd(lastAmount);
        int u=edgeData[pos].u,v=edgeData[pos].v;
        edgeData.push_back((EdgeData){u,v,d[u]++});
        edges[i]=(Edges){u,v,(unsigned)d[u]};
    }
    insert(edges,batchSize,nd,ndD,d,sizeManager);
    delete[] edges;
}
void deleteGraph(){
    Deleted *edges=new Deleted[batchSize];
    for(int i=0;i<batchSize;++i){
        int pos=r.rd(edgeData.size());
        while(edgeData[pos].nodeIdu==-1)pos=r.rd(edgeData.size());
        int u=edgeData[pos].u,idu=edgeData[pos].nodeIdu;
        edges[i]=(Deleted){u,idu};
        edgeData[pos].nodeIdu=-1;
    }
    deleteE(edges,batchSize,ndD);
    delete[] edges;
}
double totalTime=0;

int *rwD[GPUS];

__global__ void countGraph(int n,NodeData *ndD,int *a,double *b,double *c,double *d,double *e,int *log2){
    if(threadIdx.x!=0)return;
    a[0]=a[1]=a[2]=a[3]=0;
    log2[0]=0;
    log2[1]=1;
    for(int i=2;i<5000000;++i){log2[i]=log2[i>>1]<<1;if(log2[i]<i)log2[i]<<=1;}
    float b11=0,b12=0;
    float b31=0,b32=0;
    for(int i=1;i<=n;++i){
        int x=4;
        if(ndD[i].edgeSZ<256)x=1;
        else if(ndD[i].edgeSZ<65536)x=2;
        for(int j=0;j<LOGT;++j){
            if(ndD[i].num[j]==0)continue;if((100ll*i/n)!=(100ll*(i-1)/n)&&j==0)printf("%d %d %d\n",i,j,n);
            ++a[0];
            if(ndD[i].num[j]==1){
                ++a[1];b11+=1;b12+=1+ndD[i].edgeSZ;
                c[1]+=4+1;
                d[1]+=x+1;
                e[1]+=(log2[ndD[i].num[j]]+log2[ndD[i].edgeSZ])*4;
            }
            else if(ndD[i].num[j]*5<ndD[i].edgeSZ){
                ++a[2];b31+=ndD[i].num[j]*2+1;b32+=ndD[i].num[j]+ndD[i].edgeSZ;
                c[2]+=(log2[ndD[i].num[j]]+log2[ndD[i].num[j]+1])*4+1;
                d[2]+=(log2[ndD[i].num[j]]+log2[ndD[i].num[j]+1])*x+1;
                e[2]+=(log2[ndD[i].num[j]]+log2[ndD[i].edgeSZ])*4;
            }
            else if(ndD[i].num[j]>ndD[i].edgeSZ*0.5){
                ++a[3];
                c[3]+=1;
                d[3]+=1;
                e[3]+=(log2[ndD[i].num[j]]+log2[ndD[i].edgeSZ])*4;
            }
            else{
                c[0]+=(log2[ndD[i].num[j]]+log2[ndD[i].edgeSZ])*4+1;
                d[0]+=(log2[ndD[i].num[j]]+log2[ndD[i].edgeSZ])*x+1;
                e[0]+=(log2[ndD[i].num[j]]+log2[ndD[i].edgeSZ])*4;
            }
        }
    }
    b[1]=b11/b12;
    b[2]=b31/b32;
    for(int i=0;i<4;++i)(c[i]/=1048576)/=1024,(d[i]/=1048576)/=1024,(e[i]/=1048576)/=1024;
    
}
void countGraph(){
    int a[4]={0,0,0,0};//total one sparse dense
    int *aD;hipMalloc((void**)&aD,4*sizeof(int));
    double *bD;hipMalloc((void**)&bD,4*sizeof(double));
    double *cD;hipMalloc((void**)&cD,4*sizeof(double));
    double *dD;hipMalloc((void**)&dD,4*sizeof(double));
    double *eD;hipMalloc((void**)&eD,4*sizeof(double));
    int *log2;hipMalloc((void**)&log2,5000000*sizeof(int));
    countGraph<<<1,32>>>(n,ndD[0],aD,bD,cD,dD,eD,log2);
    hipDeviceSynchronize();
    HE(hipGetLastError());
    hipMemcpy(a,aD,4*sizeof(int),hipMemcpyDeviceToHost);
    double b[4],c[4],d[4],e[4];
    hipMemcpy(b,bD,4*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(c,cD,4*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(d,dD,4*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(e,eD,4*sizeof(double),hipMemcpyDeviceToHost);
    cout<<a[0]<<" "<<a[1]<<" "<<a[2]<<" "<<a[3]<<endl;
    cout<<"one"<<" "<<"sparse "<<"dense"<<endl;
    cout<<1.*a[1]/a[0]<<" "<<1.*a[2]/a[0]<<" "<<1.*a[3]/a[0]<<endl;
    cout<<b[1]<<" "<<b[2]<<" "<<b[3]<<endl;
    cout<<c[0]<<" "<<c[1]<<" "<<c[2]<<" "<<c[3]<<" "<<c[0]+c[1]+c[2]+c[3]<<endl;
    cout<<d[0]<<" "<<d[1]<<" "<<d[2]<<" "<<d[3]<<" "<<d[0]+d[1]+d[2]+d[3]<<endl;
    cout<<e[0]<<" "<<e[1]<<" "<<e[2]<<" "<<e[3]<<" "<<e[0]+e[1]+e[2]+e[3]<<endl;
}

void randomWalk(){
    //int *rw=new int[LEN*n];
    Timer tt;//tt.restart();
  //  float time=0;
//    cerr<<tt.duration()<<endl;
    tt.restart();
    #pragma omp parallel for
    for(int g=0;g<GPUS;++g){
        hipSetDevice(g);
        //if(BUFFER)resetKernel<<<BLKSZ,THDSZ>>>(n,ndD[g]);
        if(rwD[g]==NULL)hipMalloc((void **)&rwD[g],(LEN*(n/GPUS+1))*sizeof(int));
        HE(hipGetLastError());
        
   //     for(int t=0;t<2;++t){
           // GPUTimer gT;gT.init();//cerr<<g<<" "<<n<<endl;
            hipDeviceSynchronize();
            randomWalkKernel<<<BLKSZ,THDSZ>>>(g,n,ndD[g],rwD[g],((ull)new char)+g*13/*abc,bbc*/);
            hipDeviceSynchronize();
            HE(hipGetLastError());
            //randomWalkKernel<<<BLKSZ,THDSZ>>>(1,n,ndD[g],rwD[g],((ull)new char)+g*13/*abc,bbc*/);
            //hipDeviceSynchronize();
            //HE(hipGetLastError());
          //  time=max(time,gT.finish());
     //   }
    }
    totalTime+=tt.duration();//time/1000;
  /*   #pragma omp parallel for
    for(int g=0;g<GPUS;++g){
        hipSetDevice(g);
        int lPos=1+n*g/GPUS,rPos=n*(g+1)/GPUS;//(rPos-lPos+1)
        hipMemcpy(rw+(lPos-1)*LEN,rwD[g],(LEN*n/4)*sizeof(int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
   for(int i=0;i<=10;++i){
    for(int j=1;j<=80;++j)cout<<rw[i*80+j-1]<<" ";cout<<endl;}

    long long a=0,mx=0;
    for(int iii=1;iii<=1000000;++iii){
        int pos=r.rd(edgeData.size());
        //while(edgeData[pos].u<=n/2)pos=r.rd(edgeData.size());
        int u=edgeData[pos].u;
        long long sum=0;
        for(int i=0;i<LEN*n/10;++i)
        if((rw[i])==u)++sum;
        a+=sum;mx=max(mx,sum);
        if(iii%10==0)cout<<1.*a/iii<<" "<<mx<<endl;
    }
*/
    //int *xxx=new int[n];
   // memset(xxx,0,sizeof(int)*n);
//    for(int i=0;i<n*80;++i)if(rw[i]!=-1&&rw[i]!=0)++xxx[rw[i]];
    //sort(xxx,xxx+n);
   // for(int i=1;i<=3000;++i)cerr<<xxx[n-i]<<" "<<100.*xxx[n-i]/(80*n)<<"% "<<endl;
    //for(int i=1;i<=10;++i){
      //  for(int j=1;j<=LEN;++j)printf("%d ",rw[(i-1)*LEN+j-1]);
        //puts("");
   // }
   // delete[] rw;
}
int deepwalk(){
    freopen("../dataset/AM","r",stdin);
    Timer TT;TT.restart();
    omp_set_num_threads(CPUTHD);
    loadGraph();
    randomWalk();
    Timer T;T.restart();
 //   freopen("out.txt","w",stdout);
    for(int i=0;i<10;++i){
        insertGraph();
        deleteGraph();
        randomWalk();
    }
    cerr<<"Evaluation time: "<<T.duration()<<" s."<<endl;
    cerr<<"Random walk in "<<totalTime<<" s."<<endl;
    cerr<<"Total time: "<<TT.duration()<<" s."<<endl;
    return 0;
}