#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<cstring>
#include<omp.h>
#include"api.h"
#include"apps.h"
#include"gpuMem.cuh"
#include"operation.cuh"
#include"Bingo.cuh"
#include"utils.cuh"
#include"test.cuh"
using namespace std;
CPURand r;
int n,batchSize=100000;
int *d,*sizeManager;
vector<EdgeData>edgeData;
bool buffer=0;
NodeData *nd[GPUS],*ndD[GPUS];
void loadGraph(){    
    r.init();mp.init();
    int u,v;n=0;
    while(scanf("%d%d",&u,&v)!=EOF){
        EdgeData tmp;tmp.u=u+1,tmp.v=v+1;
        edgeData.push_back(tmp);
    }   
    cerr<<"Edge number: "<<edgeData.size()<<"."<<endl;
    for(int i=0;i<edgeData.size();++i)n=max(n,max(edgeData[i].u,edgeData[i].v));
    int amount=edgeData.size();
    Edges *edges=new Edges[amount];
    d=new int[n+1];
    memset(d,0,sizeof(int)*(n+1));
    for(int i=0;i<amount;++i){
        EdgeData *tmp=&edgeData[i];
        tmp->nodeIdu=d[tmp->u]++;
    }
    #pragma omp parallel for
    for(int i=0;i<amount;++i){
        int u=edgeData[i].u,v=edgeData[i].v;
        edges[i]=(Edges){u,v,(unsigned)d[v]};
    }
    build(edges,amount,nd,ndD,d,&sizeManager,n);
    delete[] edges;
}
void insertGraph(){
    Edges *edges=new Edges[batchSize];
    int lastAmount=edgeData.size();
    for(int i=0;i<batchSize;++i){
        int pos=r.rd(lastAmount);
        int u=edgeData[pos].u,v=edgeData[pos].v;
        edgeData.push_back((EdgeData){u,v,d[u]++});
        edges[i]=(Edges){u,v,(unsigned)d[u]};
    }
    insert(edges,batchSize,nd,ndD,d,sizeManager);
    delete[] edges;
}
void deleteGraph(){
    Deleted *edges=new Deleted[batchSize];
    for(int i=0;i<batchSize;++i){
        int pos=r.rd(edgeData.size());
        while(edgeData[pos].nodeIdu==-1)pos=r.rd(edgeData.size());
        int u=edgeData[pos].u,idu=edgeData[pos].nodeIdu;
        edges[i]=(Deleted){u,idu};
        edgeData[pos].nodeIdu=-1;
    }
    deleteE(edges,batchSize,ndD);
    delete[] edges;
}
double totalTime=0;

int *rwD[GPUS];

void randomWalk(){
    //int *rw=new int[LEN*n];
    Timer tt;//tt.restart();
  //  float time=0;
//    cerr<<tt.duration()<<endl;
    tt.restart();
    #pragma omp parallel for
    for(int g=0;g<GPUS;++g){
        hipSetDevice(g);
        //if(BUFFER)resetKernel<<<BLKSZ,THDSZ>>>(n,ndD[g]);
        if(rwD[g]==NULL)hipMalloc((void **)&rwD[g],(LEN*(n/GPUS+1))*sizeof(int));
        HE(hipGetLastError());
        
   //     for(int t=0;t<2;++t){
           // GPUTimer gT;gT.init();//cerr<<g<<" "<<n<<endl;
            hipDeviceSynchronize();
            randomWalkKernel<<<BLKSZ,THDSZ>>>(g,n,ndD[g],rwD[g],((ull)new char)+g*13/*abc,bbc*/);
            hipDeviceSynchronize();
            HE(hipGetLastError());
         
          //  time=max(time,gT.finish());
     //   }
    }
    totalTime+=tt.duration();//time/1000;
  /*   #pragma omp parallel for
    for(int g=0;g<GPUS;++g){
        hipSetDevice(g);
        int lPos=1+n*g/GPUS,rPos=n*(g+1)/GPUS;//(rPos-lPos+1)
        hipMemcpy(rw+(lPos-1)*LEN,rwD[g],(LEN*n/4)*sizeof(int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
   for(int i=0;i<=10;++i){
    for(int j=1;j<=80;++j)cout<<rw[i*80+j-1]<<" ";cout<<endl;}


*/

   // delete[] rw;
}
int deepwalk(){
    freopen("../dataset/AM","r",stdin);
    Timer TT;TT.restart();
    omp_set_num_threads(CPUTHD);
    loadGraph();
    randomWalk();
    Timer T;T.restart();
 //   freopen("out.txt","w",stdout);
    for(int i=0;i<10;++i){
        insertGraph();
        deleteGraph();
        randomWalk();
    }
    cerr<<"Evaluation time: "<<T.duration()<<" s."<<endl;
    cerr<<"Random walk in "<<totalTime<<" s."<<endl;
    cerr<<"Total time: "<<TT.duration()<<" s."<<endl;
    return 0;
}